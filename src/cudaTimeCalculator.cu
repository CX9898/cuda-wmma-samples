#include "cudaTimeCalculator.cuh"
#include "cudaErrorCheck.cuh"

cudaTimeCalculator::cudaTimeCalculator(){
    time = 0.0f;

    cudaErrCheck(hipEventCreate(&star));
    cudaErrCheck(hipEventCreate(&stop));
}

cudaTimeCalculator::~cudaTimeCalculator() {
    cudaErrCheck(hipEventDestroy(star));
    cudaErrCheck(hipEventDestroy(stop));
}

void cudaTimeCalculator::startClock() {
    cudaErrCheck(hipEventRecord(star));
}

void cudaTimeCalculator::endClock() {
    cudaErrCheck(hipEventRecord(stop));
    cudaErrCheck(hipEventSynchronize(stop));
}

float cudaTimeCalculator::getTime() {
    cudaErrCheck(hipEventElapsedTime(&time, star, stop));
    return time;
}