#include "cudaTimeCalculator.cuh"
#include "cudaErrorCheck.cuh"

cudaTimeCalculator::cudaTimeCalculator(){
    time_ = 0.0f;

    cudaErrCheck(hipEventCreate(&star_));
    cudaErrCheck(hipEventCreate(&stop_));
}

cudaTimeCalculator::~cudaTimeCalculator() {
    cudaErrCheck(hipEventDestroy(star_));
    cudaErrCheck(hipEventDestroy(stop_));
}

void cudaTimeCalculator::startClock() {
    cudaErrCheck(hipEventRecord(star_));
}

void cudaTimeCalculator::endClock() {
    cudaErrCheck(hipEventRecord(stop_));
    cudaErrCheck(hipEventSynchronize(stop_));
}

float cudaTimeCalculator::getTime() {
    cudaErrCheck(hipEventElapsedTime(&time_, star_, stop_));
    return time_;
}