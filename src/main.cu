#include <cstdio>

#include <hiprand.h>
#include <hipblas.h>

#include "kernelFunc.cuh"
#include "hostFunc.hpp"
#include "cudaErrorCheck.cuh"
#include "matrixSetting.hpp"

int main() {
    float *aFp32;
    float *bFp32;

    half *aFp16;
    half *bFp16;

    float *cMmaExampleCommon;
    float *cCublasGemmEx;
    float *cWmmaExample1DGrid;
    float *cWmmaExample2DGrid;
    float *cWmmaExample2DGrid2;
    float *cWmmaExample2DGrid3;

    const float alpha = 2.0f;
    const float beta = 2.0f;

    // Allocated memory in the global memory of the GPU
    {
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&aFp32), MATRIX_A_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&bFp32), MATRIX_B_SIZE * sizeof(float)));

        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&aFp16), MATRIX_A_SIZE * sizeof(half)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&bFp16), MATRIX_B_SIZE * sizeof(half)));

        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cMmaExampleCommon), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cCublasGemmEx), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample1DGrid), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample2DGrid), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample2DGrid2), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample2DGrid3), MATRIX_C_SIZE * sizeof(float)));
    }

    // using cuRAND to initialize
    {
        hiprandGenerator_t curandGen;

        curandErrCheck(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(curandGen, aFp32, MATRIX_A_SIZE));
        curandErrCheck(hiprandGenerateUniform(curandGen, bFp32, MATRIX_B_SIZE));

        const int numThreadPerBlock = 256;
        convertFp32ToFp16<<< (MATRIX_A_SIZE + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            MATRIX_A_SIZE, aFp32, aFp16);
        convertFp32ToFp16<<< (MATRIX_B_SIZE + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            MATRIX_B_SIZE, bFp32, bFp16);

        float *c;
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&c), MATRIX_C_SIZE * sizeof(float)));
        curandErrCheck(hiprandGenerateUniform(curandGen, c, MATRIX_C_SIZE));

        cudaErrCheck(hipMemcpy(cMmaExampleCommon, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cCublasGemmEx, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample1DGrid, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample2DGrid, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample2DGrid2, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample2DGrid3, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));

        curandErrCheck(hiprandDestroyGenerator(curandGen));
    }

    // using mmaExampleCommon computation
    {
        const int numThreadPerBlocks = 1024;
        const int numBlocks = (MATRIX_C_SIZE + numThreadPerBlocks - 1) / numThreadPerBlocks;
        mmaExampleCommon<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                            alpha, beta,
                                                            aFp16, bFp16, cMmaExampleCommon);
    }

    // using cuBLAS computation
    {
        printf("---------------------------\n");
        printf("Running with cuBLAS...\n");

        hipEvent_t startCublas;
        hipEvent_t stopCublas;

        cudaErrCheck(hipEventCreate(&startCublas));
        cudaErrCheck(hipEventCreate(&stopCublas));

        hipblasHandle_t cublasHandle;
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipEventRecord(startCublas));
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                                    MATRIX_M, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    aFp16, HIP_R_16F, MATRIX_M,
                                    bFp16, HIP_R_16F, MATRIX_K,
                                    &beta,
                                    cCublasGemmEx, HIP_R_32F, MATRIX_M,
                                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        cudaErrCheck(hipEventRecord(stopCublas));
        cudaErrCheck(hipEventSynchronize(stopCublas));

        float cublasTime;
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startCublas, stopCublas));
        printf("hipblasGemmEx time : %fms\n", cublasTime);

        cublasErrCheck(hipblasDestroy(cublasHandle));

        cudaErrCheck(hipEventDestroy(startCublas));
        cudaErrCheck(hipEventDestroy(stopCublas));
    }

    // using wmmaExample1DGrid computation
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample1DGrid...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        const int wmmaCalculatesOneResultTileSize = WMMA_M * WMMA_N;
        int numThreadPerBlocks = WARP_SIZE * 1;
        int numBlocks = (MATRIX_C_SIZE / wmmaCalculatesOneResultTileSize * WARP_SIZE + numThreadPerBlocks - 1)
            / numThreadPerBlocks;

        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmmaExample1DGrid<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                             alpha, beta,
                                                             aFp16, bFp16, cWmmaExample1DGrid);
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmmaExample1DGrid time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    // using wmmaExample2DGrid computation
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample2DGrid...\n");

        hipEvent_t start;
        hipEvent_t stop;

        cudaErrCheck(hipEventCreate(&start));
        cudaErrCheck(hipEventCreate(&stop));

        dim3 gridDim;
        dim3 blockDim;

        blockDim.x = WARP_SIZE;
        blockDim.y = WARP_SIZE;

        const int numCountRowOfOutputMatrixPerBlock = (int) (WMMA_M * blockDim.x / 32);
        const int numCountColOfOutputMatrixPerBlock = (int) (WMMA_N * blockDim.y);
        gridDim.x = (MATRIX_M + numCountRowOfOutputMatrixPerBlock - 1) / numCountRowOfOutputMatrixPerBlock;
        gridDim.y = (MATRIX_N + numCountColOfOutputMatrixPerBlock - 1) / numCountColOfOutputMatrixPerBlock;

        cudaErrCheck(hipEventRecord(start));
        wmmaExample2DGrid<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                 alpha, beta,
                                                 aFp16, bFp16, cWmmaExample2DGrid);
        cudaErrCheck(hipEventRecord(stop));
        cudaErrCheck(hipEventSynchronize(stop));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, start, stop));
        printf("wmmaExample2DGrid time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(start));
        cudaErrCheck(hipEventDestroy(stop));
    }

    // using wmmaExample2DGrid2 computation
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample2DGrid2...\n");

        hipEvent_t start2;
        hipEvent_t stop2;

        cudaErrCheck(hipEventCreate(&start2));
        cudaErrCheck(hipEventCreate(&stop2));

        dim3 gridDim;
        dim3 blockDim;

        blockDim.x = WARP_SIZE;
        blockDim.y = WARP_SIZE;

        const int numCountRowOfOutputMatrixPerBlock = (int) (WMMA_M * blockDim.x / 32);
        const int numCountColOfOutputMatrixPerBlock = (int) (WMMA_N * blockDim.y);
        gridDim.x = (MATRIX_M + numCountRowOfOutputMatrixPerBlock - 1) / numCountRowOfOutputMatrixPerBlock;
        gridDim.y = (MATRIX_N + numCountColOfOutputMatrixPerBlock - 1) / numCountColOfOutputMatrixPerBlock;

        cudaErrCheck(hipEventRecord(start2));
        wmmaExample2DGrid2<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                  alpha, beta,
                                                  aFp16, bFp16, cWmmaExample2DGrid2);

        cudaErrCheck(hipEventRecord(stop2));
        cudaErrCheck(hipEventSynchronize(stop2));
        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, start2, stop2));
        printf("wmmaExample2DGrid2 time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(start2));
        cudaErrCheck(hipEventDestroy(stop2));
    }

    // using wmmaExample2DGrid3 computation
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample2DGrid3...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        dim3 gridDim;
        dim3 blockDim;

        // blockDim.x must be a multiple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmmaExample2DGrid3<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                  alpha, beta,
                                                  aFp16, bFp16, cWmmaExample2DGrid3);
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmmaExample2DGrid3 time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    if (!checkDevData(MATRIX_C_SIZE, cCublasGemmEx, cWmmaExample2DGrid3)) {
        printf("Error! Function hipblasGemmEx, wmmaExample2DGrid3 Check no passes!\n");
    } else {
        printf("Function hipblasGemmEx, wmmaExample2DGrid3 Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cMmaExampleCommon, cWmmaExample1DGrid)) {
        printf("Error! Function mmaExampleCommon, wmmaExample1DGrid Check no passes!\n");
    } else {
        printf("Function mmaExampleCommon, wmmaExample1DGrid Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cMmaExampleCommon, cWmmaExample2DGrid)) {
        printf("Error! Function mmaExampleCommon, wmmaExample2DGrid Check no passes!\n");
    } else {
        printf("Function mmaExampleCommon, wmmaExample2DGrid Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cWmmaExample1DGrid, cWmmaExample2DGrid)) {
        printf("Error! Function wmmaExample1DGrid, wmmaExample2DGrid Check no passes!\n");
    } else {
        printf("Function wmmaExample1DGrid, wmmaExample2DGrid Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cCublasGemmEx, cWmmaExample2DGrid2)) {
        printf("Error! Function hipblasGemmEx, wmmaExample2DGrid2 Check no passes!\n");
    } else {
        printf("Function hipblasGemmEx, wmmaExample2DGrid2 Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cWmmaExample2DGrid, cWmmaExample2DGrid2)) {
        printf("Error! Function wmmaExample2DGrid, wmmaExample2DGrid2 Check no passes!\n");
    } else {
        printf("Function wmmaExample2DGrid, wmmaExample2DGrid2 Check passes!\n");
    }

    cudaErrCheck(hipFree(aFp32));
    cudaErrCheck(hipFree(bFp32));
    cudaErrCheck(hipFree(aFp16));
    cudaErrCheck(hipFree(bFp16));
    cudaErrCheck(hipFree(cCublasGemmEx));
    cudaErrCheck(hipFree(cWmmaExample2DGrid3));
    cudaErrCheck(hipFree(cWmmaExample1DGrid));
    cudaErrCheck(hipFree(cWmmaExample2DGrid));
    cudaErrCheck(hipFree(cWmmaExample2DGrid2));

    return 0;
}