#include "kernelFunc.cuh"
#include "hostFunc.hpp"
#include "cudaErrorCheck.hpp"

int main() {
    float *aFp32;
    float *bFp32;

    half *aFp16;
    half *bFp16;

    float *cWmmaExampleCommon;
    float *cCublasGemmEx;
    float *cWmmaExample1DGrid;
    float *cWmmaExample2DGrid;
    float *cWmma_example;

    const float alpha = 2.0f;
    const float beta = 2.0f;

    const int numMatrixADates = MATRIX_M * MATRIX_K;
    const int numMatrixBDates = MATRIX_K * MATRIX_N;
    const int numMatrixCDates = MATRIX_M * MATRIX_N;

    /* Allocated memory in the global memory of the GPU */
    {
        cudaErrCheck(hipMalloc((void **) &aFp32, numMatrixADates * sizeof(float)));
        cudaErrCheck(hipMalloc((void **) &bFp32, numMatrixBDates * sizeof(float)));

        cudaErrCheck(hipMalloc((void **) &aFp16, numMatrixADates * sizeof(half)));
        cudaErrCheck(hipMalloc((void **) &bFp16, numMatrixBDates * sizeof(half)));

        cudaErrCheck(hipMalloc((void **) &cWmmaExampleCommon, numMatrixCDates * sizeof(float)));
        cudaErrCheck(hipMalloc((void **) &cCublasGemmEx, numMatrixCDates * sizeof(float)));
        cudaErrCheck(hipMalloc((void **) &cWmmaExample1DGrid, numMatrixCDates * sizeof(float)));
        cudaErrCheck(hipMalloc((void **) &cWmmaExample2DGrid, numMatrixCDates * sizeof(float)));
        cudaErrCheck(hipMalloc((void **) &cWmma_example, numMatrixCDates * sizeof(float)));
    }

    /* using hiprand to initialize */
    {
        hiprandGenerator_t curandGen;

        curandErrCheck(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(curandGen, aFp32, numMatrixADates));
        curandErrCheck(hiprandGenerateUniform(curandGen, bFp32, numMatrixBDates));

        float *c;
        cudaErrCheck(hipMalloc((void **) &c, numMatrixCDates * sizeof(float)));
        curandErrCheck(hiprandGenerateUniform(curandGen, c, numMatrixCDates));

        cudaErrCheck(hipMemcpy(cWmmaExampleCommon, c, numMatrixCDates, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cCublasGemmEx, c, numMatrixCDates, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample1DGrid, c, numMatrixCDates, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample2DGrid, c, numMatrixCDates, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmma_example, c, numMatrixCDates, hipMemcpyDeviceToDevice));

        curandErrCheck(hiprandDestroyGenerator(curandGen));

        const int numThreadPerBlock = 256;
        convertFp32ToFp16<<< (numMatrixADates + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            aFp16, aFp32, numMatrixADates);
        convertFp32ToFp16<<< (numMatrixBDates + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            bFp16, bFp32, numMatrixBDates);
    }

//    std::vector<float> aHost(numMatrixADates);
//    std::vector<float> bHost(numMatrixBDates);
//    std::vector<float> cHost(numMatrixCDates);
//
//    hipMemcpy(aHost.data(), aFp16, numMatrixADates, hipMemcpyDeviceToHost);
//    hipMemcpy(bHost.data(), bFp16, numMatrixBDates, hipMemcpyDeviceToHost);
//    hipMemcpy(cHost.data(), cWmmaExample1DGrid, numMatrixCDates, hipMemcpyDeviceToHost);
//
//    mmaHost(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aHost, bHost, cHost);

    /* using mmaExampleCommon computation  */
    {
        const int numThreadPerBlocks = 1024;
        const int numBlocks = (numMatrixCDates + numThreadPerBlocks - 1) / numThreadPerBlocks;
        mmaExampleCommon<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                            alpha, beta,
                                                            aFp16, bFp16, cWmmaExampleCommon);
    }

    /* using cuBLAS computation */
    {
        printf("---------------------------\n");
        printf("Running with cuBLAS...\n");

        hipEvent_t startCublas;
        hipEvent_t stopCublas;

        cudaErrCheck(hipEventCreate(&startCublas));
        cudaErrCheck(hipEventCreate(&stopCublas));

        hipblasHandle_t cublasHandle;
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipEventRecord(startCublas));
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    MATRIX_M, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    aFp16, HIP_R_16F, MATRIX_M,
                                    bFp16, HIP_R_16F, MATRIX_K,
                                    &beta,
                                    cCublasGemmEx, HIP_R_32F, MATRIX_M,
                                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        cudaErrCheck(hipEventRecord(stopCublas));
        cudaErrCheck(hipEventSynchronize(stopCublas));

        float cublasTime;
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startCublas, stopCublas));
        printf("hipblasGemmEx time : %fms\n", cublasTime);

        cublasErrCheck(hipblasDestroy(cublasHandle));

        cudaErrCheck(hipEventDestroy(startCublas));
        cudaErrCheck(hipEventDestroy(stopCublas));
    }

    /* using wmma-example computation */
    {
        printf("---------------------------\n");
        printf("Running with wmma-example...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        dim3 gridDim;
        dim3 blockDim;

        // blockDim.x must be a multiple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
        printf("gridDim.x = %d gridDim.y = %d blockDim.x = %d blockDim.y = %d\n",
               gridDim.x, gridDim.y,
               blockDim.x, blockDim.y);
        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmma_example<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aFp16, bFp16, cWmma_example);
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmma_example time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    /* using wmmaExample1DGrid computation */
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample1DGrid...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        const int wmmaCalculatesOneResultTileSize = WMMA_M * WMMA_N;
        int numThreadPerBlocks = WARP_SIZE * 1;
        int numBlocks = (numMatrixCDates / wmmaCalculatesOneResultTileSize * WARP_SIZE + numThreadPerBlocks - 1)
            / numThreadPerBlocks;
        printf("numBlocks = %d numThreadPerBlocks = %d\n", numBlocks, numThreadPerBlocks);
        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmmaExample1DGrid<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                             alpha, beta,
                                                             aFp16, bFp16, cWmmaExample1DGrid);
        printf("%s\n", hipGetErrorString(hipGetLastError()));
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmmaExample1DGrid time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    /* using wmmaExample2DGrid computation */
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample2DGrid...\n");

        hipEvent_t start;
        hipEvent_t stop;

        cudaErrCheck(hipEventCreate(&start));
        cudaErrCheck(hipEventCreate(&stop));

        dim3 gridDim;
        dim3 blockDim;

        blockDim.x = WARP_SIZE;
        blockDim.y = WARP_SIZE;

        const int numCountRowOfOutputMatrixPerBlock = (WMMA_M * blockDim.x / 32);
        const int numCountColOfOutputMatrixPerBlock = (WMMA_N * blockDim.y);
        gridDim.x = (MATRIX_M + numCountRowOfOutputMatrixPerBlock - 1) / numCountRowOfOutputMatrixPerBlock;
        gridDim.y = (MATRIX_N + numCountColOfOutputMatrixPerBlock - 1) / numCountColOfOutputMatrixPerBlock;

        cudaErrCheck(hipEventRecord(start));
        wmmaExample2DGrid<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                 alpha, beta,
                                                 aFp16, bFp16, cWmmaExample2DGrid);
        printf("%s\n", hipGetErrorString(hipGetLastError()));
        cudaErrCheck(hipEventRecord(stop));
        cudaErrCheck(hipEventSynchronize(stop));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, start, stop));
        printf("wmmaExample2DGrid time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(start));
        cudaErrCheck(hipEventDestroy(stop));
    }

    if (!checkDevData(numMatrixCDates, cCublasGemmEx, cWmmaExample1DGrid)) {
        printf("Error! cublas, wmmaExample1DGrid Check no passes!\n");
    } else {
        printf("cublas, wmmaExample1DGrid Check passes!\n");
    }

    if (!checkDevData(numMatrixCDates, cCublasGemmEx, cWmma_example)) {
        printf("Error! cublas, wmma_example Check no passes!\n");
    } else {
        printf("cublas, wmma_example Check passes!\n");
    }

    if (!checkDevData(numMatrixCDates, cWmmaExampleCommon, cWmmaExample1DGrid)) {
        printf("Error! mmaExampleCommon, wmmaExample1DGrid Check no passes!\n");
    } else {
        printf("mmaExampleCommon, wmmaExample1DGrid Check passes!\n");
    }

    if (!checkDevData(numMatrixCDates, cWmmaExampleCommon, cWmmaExample2DGrid)) {
        printf("Error! wmmaExampleCommon, wmmaExample2DGrid Check no passes!\n");
    } else {
        printf("wmmaExampleCommon, wmmaExample2DGrid Check passes!\n");
    }

    if (!checkDevData(numMatrixCDates, cWmmaExample1DGrid, cWmmaExample2DGrid)) {
        printf("Error! wmmaExample1DGrid, wmmaExample2DGrid Check no passes!\n");
    } else {
        printf("wmmaExample1DGrid, wmmaExample2DGrid Check passes!\n");
    }

    cudaErrCheck(hipFree(aFp32));
    cudaErrCheck(hipFree(bFp32));
    cudaErrCheck(hipFree(aFp16));
    cudaErrCheck(hipFree(bFp16));
    cudaErrCheck(hipFree(cCublasGemmEx));
    cudaErrCheck(hipFree(cWmmaExample1DGrid));
    cudaErrCheck(hipFree(cWmma_example));

    return 0;
}