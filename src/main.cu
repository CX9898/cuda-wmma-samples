
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hiprand/hiprand.h>
#include <mma.h>
#include <hipblas.h>

using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
#define MATRIX_M 16384
#define MATRIX_N 16384
#define MATRIX_K 16384

// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
    if (stat != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
    }
}

__global__ void convertFp32ToFp16(half *out, float *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx];
    }
}

__global__ void wmmaExample(const int M, const int N, const int K,
                            const float alpha, const float beta,
                            const half *mtrA, const half *mtrB, float *mtrC) {
    const int warpID = (int) (blockDim.x * blockIdx.x + threadIdx.x) / warpSize;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

}

int main() {
    float *a_fp32;
    float *b_fp32;

    half *a_fp16;
    half *b_fp16;

    float *c;
    float *c_cublas;

    const float alpha = 2.0f;
    const float beta = 2.0f;

    cudaErrCheck(hipMalloc((void **) &a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));

    cudaErrCheck(hipMalloc((void **) &a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
    cudaErrCheck(hipMalloc((void **) &b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

    cudaErrCheck(hipMalloc((void **) &c, MATRIX_M * MATRIX_N * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));

    /* using curand to initialize */
    {
        hiprandGenerator_t curandGen;

        curandErrCheck(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(curandGen, a_fp32, MATRIX_M * MATRIX_K));
        curandErrCheck(hiprandGenerateUniform(curandGen, b_fp32, MATRIX_K * MATRIX_N));
        curandErrCheck(hiprandGenerateUniform(curandGen, c, MATRIX_M * MATRIX_N));

        curandErrCheck(hiprandDestroyGenerator(curandGen));

        const int numThreadPerBlock = 256;
        const int numBlocks = (MATRIX_M * MATRIX_K + 255) / 256;
        convertFp32ToFp16<<< numBlocks, numThreadPerBlock>>>(a_fp16, a_fp32, MATRIX_M * MATRIX_K);
        convertFp32ToFp16<<< numBlocks, numThreadPerBlock>>>(b_fp16, b_fp32, MATRIX_K * MATRIX_N);
    }

    /* using cuBLAS computation */
    {
        printf("Running with cuBLAS...\n");

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));

        hipblasHandle_t cublasHandle;
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipEventRecord(startcublas));
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    MATRIX_M, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    a_fp16, HIP_R_16F, MATRIX_M,
                                    b_fp16, HIP_R_16F, MATRIX_K,
                                    &beta,
                                    c_cublas, HIP_R_32F, MATRIX_M,
                                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));

        float cublasTime;
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
        printf("hipblasGemmEx time : %fms\n", cublasTime);

        cublasErrCheck(hipblasDestroy(cublasHandle));

        cudaErrCheck(hipEventDestroy(startcublas));
        cudaErrCheck(hipEventDestroy(stopcublas));
    }

    /* using wmmaExample computation */
    {
        printf("Running with wmmaExample...\n");

        hipEvent_t startWMMAEx;
        hipEvent_t stopWMMAEx;

        cudaErrCheck(hipEventCreate(&startWMMAEx));
        cudaErrCheck(hipEventCreate(&stopWMMAEx));

        dim3 gridDim;
        dim3 blockDim;

        cudaErrCheck(hipEventRecord(startWMMAEx));
        wmmaExample<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, a_fp16, b_fp16, c);
        cudaErrCheck(hipEventRecord(stopWMMAEx));
        cudaErrCheck(hipEventSynchronize(stopWMMAEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMAEx, stopWMMAEx));
        printf("wmmaExample time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWMMAEx));
        cudaErrCheck(hipEventDestroy(stopWMMAEx));
    }

    cudaErrCheck(hipFree(a_fp32));
    cudaErrCheck(hipFree(b_fp32));
    cudaErrCheck(hipFree(a_fp16));
    cudaErrCheck(hipFree(b_fp16));
    cudaErrCheck(hipFree(c));
    cudaErrCheck(hipFree(c_cublas));

    return 0;
}