#include <cstdio>

#include <hiprand.h>
#include <hipblas.h>

#include "kernelFunc.cuh"
#include "hostFunc.hpp"
#include "cudaErrorCheck.hpp"
#include "matrixSetting.hpp"

int main() {
    float *aFp32;
    float *bFp32;

    half *aFp16;
    half *bFp16;

    float *cMmaExampleCommon;
    float *cCublasGemmEx;
    float *cWmmaExample1DGrid;
    float *cWmmaExample2DGrid;
    float *cWmma_example;

    const float alpha = 2.0f;
    const float beta = 2.0f;

    /* Allocated memory in the global memory of the GPU */
    {
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&aFp32), MATRIX_A_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&bFp32), MATRIX_B_SIZE * sizeof(float)));

        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&aFp16), MATRIX_A_SIZE * sizeof(half)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&bFp16), MATRIX_B_SIZE * sizeof(half)));

        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cMmaExampleCommon), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cCublasGemmEx), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample1DGrid), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample2DGrid), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmma_example), MATRIX_C_SIZE * sizeof(float)));
    }

    /* using hiprand to initialize */
    {
        hiprandGenerator_t curandGen;

        curandErrCheck(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(curandGen, aFp32, MATRIX_A_SIZE));
        curandErrCheck(hiprandGenerateUniform(curandGen, bFp32, MATRIX_B_SIZE));

        const int numThreadPerBlock = 256;
        convertFp32ToFp16<<< (MATRIX_A_SIZE + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            aFp16, aFp32, MATRIX_A_SIZE);
        convertFp32ToFp16<<< (MATRIX_B_SIZE + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            bFp16, bFp32, MATRIX_B_SIZE);

        float *c;
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&c), MATRIX_C_SIZE * sizeof(float)));
        curandErrCheck(hiprandGenerateUniform(curandGen, c, MATRIX_C_SIZE));

        cudaErrCheck(hipMemcpy(cMmaExampleCommon, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cCublasGemmEx, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample1DGrid, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample2DGrid, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmma_example, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));

        curandErrCheck(hiprandDestroyGenerator(curandGen));
    }

//    std::vector<float> aHost(MATRIX_A_SIZE);
//    std::vector<float> bHost(MATRIX_B_SIZE);
//    std::vector<float> cHost(MATRIX_C_SIZE);
//
//    hipMemcpy(aHost.data(), aFp16, MATRIX_A_SIZE, hipMemcpyDeviceToHost);
//    hipMemcpy(bHost.data(), bFp16, MATRIX_B_SIZE, hipMemcpyDeviceToHost);
//    hipMemcpy(cHost.data(), cWmmaExample1DGrid, MATRIX_C_SIZE, hipMemcpyDeviceToHost);
//
//    mmaHost(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aHost, bHost, cHost);

    /* using mmaExampleCommon computation  */
    {
        const int numThreadPerBlocks = 1024;
        const int numBlocks = (MATRIX_C_SIZE + numThreadPerBlocks - 1) / numThreadPerBlocks;
        mmaExampleCommon<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                            alpha, beta,
                                                            aFp16, bFp16, cMmaExampleCommon);
    }

    /* using cuBLAS computation */
    {
        printf("---------------------------\n");
        printf("Running with cuBLAS...\n");

        hipEvent_t startCublas;
        hipEvent_t stopCublas;

        cudaErrCheck(hipEventCreate(&startCublas));
        cudaErrCheck(hipEventCreate(&stopCublas));

        hipblasHandle_t cublasHandle;
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipEventRecord(startCublas));
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    MATRIX_M, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    aFp16, HIP_R_16F, MATRIX_M,
                                    bFp16, HIP_R_16F, MATRIX_K,
                                    &beta,
                                    cCublasGemmEx, HIP_R_32F, MATRIX_M,
                                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        cudaErrCheck(hipEventRecord(stopCublas));
        cudaErrCheck(hipEventSynchronize(stopCublas));

        float cublasTime;
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startCublas, stopCublas));
        printf("hipblasGemmEx time : %fms\n", cublasTime);

        cublasErrCheck(hipblasDestroy(cublasHandle));

        cudaErrCheck(hipEventDestroy(startCublas));
        cudaErrCheck(hipEventDestroy(stopCublas));
    }

    /* using wmma-example computation */
    {
        printf("---------------------------\n");
        printf("Running with wmma-example...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        dim3 gridDim;
        dim3 blockDim;

        // blockDim.x must be a multiple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
        printf("gridDim.x = %d gridDim.y = %d blockDim.x = %d blockDim.y = %d\n",
               gridDim.x, gridDim.y,
               blockDim.x, blockDim.y);
        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmma_example<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aFp16, bFp16, cWmma_example);
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmma_example time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    /* using wmmaExample1DGrid computation */
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample1DGrid...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        const int wmmaCalculatesOneResultTileSize = WMMA_M * WMMA_N;
        int numThreadPerBlocks = WARP_SIZE * 1;
        int numBlocks = (MATRIX_C_SIZE / wmmaCalculatesOneResultTileSize * WARP_SIZE + numThreadPerBlocks - 1)
            / numThreadPerBlocks;
        printf("numBlocks = %d numThreadPerBlocks = %d\n", numBlocks, numThreadPerBlocks);
        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmmaExample1DGrid<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                             alpha, beta,
                                                             aFp16, bFp16, cWmmaExample1DGrid);
        printf("%s\n", hipGetErrorString(hipGetLastError()));
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmmaExample1DGrid time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    /* using wmmaExample2DGrid computation */
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample2DGrid...\n");

        hipEvent_t start;
        hipEvent_t stop;

        cudaErrCheck(hipEventCreate(&start));
        cudaErrCheck(hipEventCreate(&stop));

        dim3 gridDim;
        dim3 blockDim;

        blockDim.x = WARP_SIZE;
        blockDim.y = WARP_SIZE;

        const int numCountRowOfOutputMatrixPerBlock = (WMMA_M * blockDim.x / 32);
        const int numCountColOfOutputMatrixPerBlock = (WMMA_N * blockDim.y);
        gridDim.x = (MATRIX_M + numCountRowOfOutputMatrixPerBlock - 1) / numCountRowOfOutputMatrixPerBlock;
        gridDim.y = (MATRIX_N + numCountColOfOutputMatrixPerBlock - 1) / numCountColOfOutputMatrixPerBlock;

        cudaErrCheck(hipEventRecord(start));
        wmmaExample2DGrid<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                 alpha, beta,
                                                 aFp16, bFp16, cWmmaExample2DGrid);
        printf("%s\n", hipGetErrorString(hipGetLastError()));
        cudaErrCheck(hipEventRecord(stop));
        cudaErrCheck(hipEventSynchronize(stop));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, start, stop));
        printf("wmmaExample2DGrid time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(start));
        cudaErrCheck(hipEventDestroy(stop));
    }

    if (!checkDevData(MATRIX_C_SIZE, cCublasGemmEx, cWmma_example)) {
        printf("Error! hipblasGemmEx, wmma_example Check no passes!\n");
    } else {
        printf("hipblasGemmEx, wmma_example Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cMmaExampleCommon, cWmmaExample1DGrid)) {
        printf("Error! mmaExampleCommon, wmmaExample1DGrid Check no passes!\n");
    } else {
        printf("mmaExampleCommon, wmmaExample1DGrid Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cMmaExampleCommon, cWmmaExample2DGrid)) {
        printf("Error! mmaExampleCommon, wmmaExample2DGrid Check no passes!\n");
    } else {
        printf("mmaExampleCommon, wmmaExample2DGrid Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cWmmaExample1DGrid, cWmmaExample2DGrid)) {
        printf("Error! wmmaExample1DGrid, wmmaExample2DGrid Check no passes!\n");
    } else {
        printf("wmmaExample1DGrid, wmmaExample2DGrid Check passes!\n");
    }

    cudaErrCheck(hipFree(aFp32));
    cudaErrCheck(hipFree(bFp32));
    cudaErrCheck(hipFree(aFp16));
    cudaErrCheck(hipFree(bFp16));
    cudaErrCheck(hipFree(cCublasGemmEx));
    cudaErrCheck(hipFree(cWmmaExample1DGrid));
    cudaErrCheck(hipFree(cWmma_example));

    return 0;
}