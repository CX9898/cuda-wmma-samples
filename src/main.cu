#include "kernel.cuh"
#include "host.hpp"
#include "cudaErrorCheck.hpp"

int main() {
    float *aFp32;
    float *bFp32;

    half *aFp16;
    half *bFp16;

    float *cCublas;
    float *cWmmaEx;
    float *cWmmaEx2;

    const float alpha = 1.0f;
    const float beta = 1.0f;

    const int numMatrixADates = MATRIX_M * MATRIX_K;
    const int numMatrixBDates = MATRIX_K * MATRIX_N;
    const int numMatrixCDates = MATRIX_M * MATRIX_N;

    cudaErrCheck(hipMalloc((void **) &aFp32, numMatrixADates * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &bFp32, numMatrixBDates * sizeof(float)));

    cudaErrCheck(hipMalloc((void **) &aFp16, numMatrixADates * sizeof(half)));
    cudaErrCheck(hipMalloc((void **) &bFp16, numMatrixBDates * sizeof(half)));

    cudaErrCheck(hipMalloc((void **) &cCublas, numMatrixCDates * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &cWmmaEx, numMatrixCDates * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &cWmmaEx2, numMatrixCDates * sizeof(float)));

    /* using hiprand to initialize */
    {
        hiprandGenerator_t curandGen;

        curandErrCheck(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(curandGen, aFp32, numMatrixADates));
        curandErrCheck(hiprandGenerateUniform(curandGen, bFp32, numMatrixBDates));

        float *c;
        cudaErrCheck(hipMalloc((void **) &c, numMatrixCDates * sizeof(float)));
        curandErrCheck(hiprandGenerateUniform(curandGen, c, numMatrixCDates));

        cudaErrCheck(hipMemcpy(cCublas, c, numMatrixCDates, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaEx, c, numMatrixCDates, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaEx2, c, numMatrixCDates, hipMemcpyDeviceToDevice));

        curandErrCheck(hiprandDestroyGenerator(curandGen));

        const int numThreadPerBlock = 256;
        convertFp32ToFp16<<< (numMatrixADates + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            aFp16, aFp32, numMatrixADates);
        convertFp32ToFp16<<< (numMatrixBDates + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            bFp16, bFp32, numMatrixBDates);
    }

    std::vector<float> aHost(numMatrixADates);
    std::vector<float> bHost(numMatrixBDates);
    std::vector<float> cHost(numMatrixCDates);

    hipMemcpy(aHost.data(), aFp32, numMatrixADates, hipMemcpyDeviceToHost);
    hipMemcpy(bHost.data(), bFp32, numMatrixBDates, hipMemcpyDeviceToHost);
    hipMemcpy(cHost.data(), cWmmaEx, numMatrixCDates, hipMemcpyDeviceToHost);

    mmaHost(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aHost, bHost, cHost);

    /* using cuBLAS computation */
    {
        printf("---------------------------\n");
        printf("Running with cuBLAS...\n");

        hipEvent_t startCublas;
        hipEvent_t stopCublas;

        cudaErrCheck(hipEventCreate(&startCublas));
        cudaErrCheck(hipEventCreate(&stopCublas));

        hipblasHandle_t cublasHandle;
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipEventRecord(startCublas));
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    MATRIX_M, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    aFp16, HIP_R_16F, MATRIX_M,
                                    bFp16, HIP_R_16F, MATRIX_K,
                                    &beta,
                                    cCublas, HIP_R_32F, MATRIX_M,
                                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        cudaErrCheck(hipEventRecord(stopCublas));
        cudaErrCheck(hipEventSynchronize(stopCublas));

        float cublasTime;
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startCublas, stopCublas));
        printf("hipblasGemmEx time : %fms\n", cublasTime);

        cublasErrCheck(hipblasDestroy(cublasHandle));

        cudaErrCheck(hipEventDestroy(startCublas));
        cudaErrCheck(hipEventDestroy(stopCublas));
    }

    /* using wmma-example computation */
    {
        printf("---------------------------\n");
        printf("Running with wmma-example...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        dim3 gridDim;
        dim3 blockDim;

        // blockDim.x must be a multiple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
        printf("gridDim.x = %d gridDim.y = %d blockDim.x = %d blockDim.y = %d\n",
               gridDim.x, gridDim.y,
               blockDim.x, blockDim.y);
        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmma_example<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aFp16, bFp16, cWmmaEx2);
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmma_example time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    /* using wmmaExample computation */
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        const int wmmaCalculatesOneResultTileSize = WMMA_M * WMMA_N;
        int numThreadPerBlocks = WARP_SIZE * 1;
        int numBlocks = (numMatrixCDates / wmmaCalculatesOneResultTileSize * WARP_SIZE + numThreadPerBlocks - 1)
            / numThreadPerBlocks;
        printf("numBlocks = %d numThreadPerBlocks = %d\n", numBlocks, numThreadPerBlocks);
        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmmaExample<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                       alpha, beta,
                                                       aFp16, bFp16, cWmmaEx);
        printf("%s\n", hipGetErrorString(hipGetLastError()));
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmmaExample time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    if (!checkDevData(numMatrixCDates, cCublas, cWmmaEx)) {
        printf("Error! cublas, wmmaExample Check no passes!\n");
    } else {
        printf("cublas, wmmaExample Check passes!\n");
    }

    if (!checkDevData(numMatrixCDates, cCublas, cWmmaEx2)) {
        printf("Error! cublas, wmma_example Check no passes!\n");
    } else {
        printf("cublas, wmma_example Check passes!\n");
    }

    if (!checkData(numMatrixCDates, cHost, cWmmaEx)) {
        printf("Error! mmaHost, wmmaExample Check no passes!\n");
    } else {
        printf("mmaHost, wmmaExample Check passes!\n");
    }

    cudaErrCheck(hipFree(aFp32));
    cudaErrCheck(hipFree(bFp32));
    cudaErrCheck(hipFree(aFp16));
    cudaErrCheck(hipFree(bFp16));
    cudaErrCheck(hipFree(cCublas));
    cudaErrCheck(hipFree(cWmmaEx));
    cudaErrCheck(hipFree(cWmmaEx2));

    return 0;
}