#include <cstdio>

#include <hiprand.h>
#include <hipblas.h>

#include "kernelFunc.cuh"
#include "hostFunc.hpp"
#include "cudaErrorCheck.cuh"
#include "matrixSetting.hpp"
#include "cudaTimeCalculator.cuh"

int main() {
    float *aFp32;
    float *bFp32;

    half *aFp16;
    half *bFp16;

    float *cMmaExampleCommon;
    float *cCublasGemmEx;
    float *cWmmaExample1DGrid;
    float *cWmmaExample2DGrid;
    float *cWmmaExample2DGrid2;
    float *cWmmaExample2DGrid3;

    const float alpha = 2.0f;
    const float beta = 2.0f;

    // Allocated memory in the global memory of the GPU
    {
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&aFp32), MATRIX_A_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&bFp32), MATRIX_B_SIZE * sizeof(float)));

        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&aFp16), MATRIX_A_SIZE * sizeof(half)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&bFp16), MATRIX_B_SIZE * sizeof(half)));

        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cMmaExampleCommon), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cCublasGemmEx), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample1DGrid), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample2DGrid), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample2DGrid2), MATRIX_C_SIZE * sizeof(float)));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&cWmmaExample2DGrid3), MATRIX_C_SIZE * sizeof(float)));
    }

    // using cuRAND to initialize
    {
        hiprandGenerator_t curandGen;

        curandErrCheck(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(curandGen, aFp32, MATRIX_A_SIZE));
        curandErrCheck(hiprandGenerateUniform(curandGen, bFp32, MATRIX_B_SIZE));

        const int numThreadPerBlock = 256;
        convertFp32ToFp16<<< (MATRIX_A_SIZE + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            MATRIX_A_SIZE, aFp32, aFp16);
        convertFp32ToFp16<<< (MATRIX_B_SIZE + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            MATRIX_B_SIZE, bFp32, bFp16);

        float *c;
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&c), MATRIX_C_SIZE * sizeof(float)));
        curandErrCheck(hiprandGenerateUniform(curandGen, c, MATRIX_C_SIZE));

        cudaErrCheck(hipMemcpy(cMmaExampleCommon, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cCublasGemmEx, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample1DGrid, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample2DGrid, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample2DGrid2, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaExample2DGrid3, c, MATRIX_C_SIZE, hipMemcpyDeviceToDevice));

        curandErrCheck(hiprandDestroyGenerator(curandGen));
    }

    // using mmaExampleCommon computation
    {
        const int numThreadPerBlocks = 1024;
        const int numBlocks = (MATRIX_C_SIZE + numThreadPerBlocks - 1) / numThreadPerBlocks;
        mmaExampleCommon<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                            alpha, beta,
                                                            aFp16, bFp16, cMmaExampleCommon);
    }

    // using cuBLAS computation
    {
        printf("---------------------------\n");
        printf("Running with cuBLAS...\n");

        hipblasHandle_t cublasHandle;
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaTimeCalculator timeCalculator;

        timeCalculator.startClock();
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    MATRIX_M, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    aFp16, HIP_R_16F, MATRIX_M,
                                    bFp16, HIP_R_16F, MATRIX_K,
                                    &beta,
                                    cCublasGemmEx, HIP_R_32F, MATRIX_M,
                                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        timeCalculator.endClock();

        printf("hipblasGemmEx time : %fms\n", timeCalculator.getTime());

        cublasErrCheck(hipblasDestroy(cublasHandle));
    }

    // using wmmaExample1DGrid computation
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample1DGrid...\n");

        const int wmmaCalculatesOneResultTileSize = WMMA_M * WMMA_N;
        int numThreadPerBlocks = WARP_SIZE * 1;
        int numBlocks = (MATRIX_C_SIZE / wmmaCalculatesOneResultTileSize * WARP_SIZE + numThreadPerBlocks - 1)
            / numThreadPerBlocks;

        cudaTimeCalculator timeCalculator;

        timeCalculator.startClock();
        wmmaExample1DGrid<<<numBlocks, numThreadPerBlocks>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                             alpha, beta,
                                                             aFp16, bFp16, cWmmaExample1DGrid);
        timeCalculator.endClock();

        printf("wmmaExample1DGrid time : %fms\n", timeCalculator.getTime());
    }

    // using wmmaExample2DGrid computation
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample2DGrid...\n");

        dim3 gridDim;
        dim3 blockDim;

        blockDim.x = WARP_SIZE;
        blockDim.y = WARP_SIZE;

        const int numCountRowOfOutputMatrixPerBlock = (int) (WMMA_M * blockDim.x / 32);
        const int numCountColOfOutputMatrixPerBlock = (int) (WMMA_N * blockDim.y);
        gridDim.x = (MATRIX_M + numCountRowOfOutputMatrixPerBlock - 1) / numCountRowOfOutputMatrixPerBlock;
        gridDim.y = (MATRIX_N + numCountColOfOutputMatrixPerBlock - 1) / numCountColOfOutputMatrixPerBlock;

        cudaTimeCalculator timeCalculator;

        timeCalculator.startClock();
        wmmaExample2DGrid<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                 alpha, beta,
                                                 aFp16, bFp16, cWmmaExample2DGrid);
        timeCalculator.endClock();

        printf("wmmaExample2DGrid time : %fms\n", timeCalculator.getTime());
    }

    // using wmmaExample2DGrid2 computation
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample2DGrid2...\n");

        dim3 gridDim;
        dim3 blockDim;

        blockDim.x = WARP_SIZE;
        blockDim.y = WARP_SIZE;

        const int numCountRowOfOutputMatrixPerBlock = (int) (WMMA_M * blockDim.x / 32);
        const int numCountColOfOutputMatrixPerBlock = (int) (WMMA_N * blockDim.y);
        gridDim.x = (MATRIX_M + numCountRowOfOutputMatrixPerBlock - 1) / numCountRowOfOutputMatrixPerBlock;
        gridDim.y = (MATRIX_N + numCountColOfOutputMatrixPerBlock - 1) / numCountColOfOutputMatrixPerBlock;

        cudaTimeCalculator timeCalculator;

        timeCalculator.startClock();
        wmmaExample2DGrid2<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                  alpha, beta,
                                                  aFp16, bFp16, cWmmaExample2DGrid2);
        timeCalculator.endClock();

        printf("wmmaExample2DGrid2 time : %fms\n", timeCalculator.getTime());
    }

    // using wmmaExample2DGrid3 computation
    {
        printf("---------------------------\n");
        printf("Running with wmmaExample2DGrid3...\n");

        dim3 gridDim;
        dim3 blockDim;

        // blockDim.x must be a multiple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

        cudaTimeCalculator timeCalculator;

        timeCalculator.startClock();
        wmmaExample2DGrid3<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K,
                                                  alpha, beta,
                                                  aFp16, bFp16, cWmmaExample2DGrid3);
        timeCalculator.endClock();

        printf("wmmaExample2DGrid3 time : %fms\n", timeCalculator.getTime());
    }

    if (!checkDevData(MATRIX_C_SIZE, cCublasGemmEx, cWmmaExample2DGrid3)) {
        printf("Error! Function hipblasGemmEx, wmmaExample2DGrid3 Check no passes!\n");
    } else {
        printf("Function hipblasGemmEx, wmmaExample2DGrid3 Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cMmaExampleCommon, cWmmaExample1DGrid)) {
        printf("Error! Function mmaExampleCommon, wmmaExample1DGrid Check no passes!\n");
    } else {
        printf("Function mmaExampleCommon, wmmaExample1DGrid Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cMmaExampleCommon, cWmmaExample2DGrid)) {
        printf("Error! Function mmaExampleCommon, wmmaExample2DGrid Check no passes!\n");
    } else {
        printf("Function mmaExampleCommon, wmmaExample2DGrid Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cWmmaExample1DGrid, cWmmaExample2DGrid)) {
        printf("Error! Function wmmaExample1DGrid, wmmaExample2DGrid Check no passes!\n");
    } else {
        printf("Function wmmaExample1DGrid, wmmaExample2DGrid Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cCublasGemmEx, cWmmaExample2DGrid2)) {
        printf("Error! Function hipblasGemmEx, wmmaExample2DGrid2 Check no passes!\n");
    } else {
        printf("Function hipblasGemmEx, wmmaExample2DGrid2 Check passes!\n");
    }

    if (!checkDevData(MATRIX_C_SIZE, cWmmaExample2DGrid, cWmmaExample2DGrid2)) {
        printf("Error! Function wmmaExample2DGrid, wmmaExample2DGrid2 Check no passes!\n");
    } else {
        printf("Function wmmaExample2DGrid, wmmaExample2DGrid2 Check passes!\n");
    }

    cudaErrCheck(hipFree(aFp32));
    cudaErrCheck(hipFree(bFp32));
    cudaErrCheck(hipFree(aFp16));
    cudaErrCheck(hipFree(bFp16));
    cudaErrCheck(hipFree(cCublasGemmEx));
    cudaErrCheck(hipFree(cWmmaExample2DGrid3));
    cudaErrCheck(hipFree(cWmmaExample1DGrid));
    cudaErrCheck(hipFree(cWmmaExample2DGrid));
    cudaErrCheck(hipFree(cWmmaExample2DGrid2));

    return 0;
}