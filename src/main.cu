
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hiprand/hiprand.h>
#include <mma.h>
#include <hipblas.h>

using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
#define MATRIX_M 16384
#define MATRIX_N 16384
#define MATRIX_K 16384

// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
    if (stat != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
    }
}

__global__ void convertFp32ToFp16(half *out, float *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx];
    }
}

__global__ void wmmaExample(const int M, const int N, const int K,
                            const float alpha, const float beta,
                            const half *mtrA, const half *mtrB, float *mtrC) {
    const int warpID = (int) (blockDim.x * blockIdx.x + threadIdx.x) / warpSize;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

}

int main() {
    float *aFp32;
    float *bFp32;

    half *aFp16;
    half *bFp16;

    float *cWmmaEx;
    float *cCublas;

    const float alpha = 2.0f;
    const float beta = 2.0f;

    const int numMatrixADates = MATRIX_M * MATRIX_K;
    const int numMatrixBDates = MATRIX_K * MATRIX_N;
    const int numMatrixCDates = MATRIX_M * MATRIX_N;

    cudaErrCheck(hipMalloc((void **) &aFp32, numMatrixADates * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &bFp32, numMatrixBDates * sizeof(float)));

    cudaErrCheck(hipMalloc((void **) &aFp16, numMatrixADates * sizeof(half)));
    cudaErrCheck(hipMalloc((void **) &bFp16, numMatrixBDates * sizeof(half)));

    cudaErrCheck(hipMalloc((void **) &cWmmaEx, numMatrixCDates * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &cCublas, numMatrixCDates * sizeof(float)));

    /* using curand to initialize */
    {
        hiprandGenerator_t curandGen;

        curandErrCheck(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(curandGen, aFp32, numMatrixADates));
        curandErrCheck(hiprandGenerateUniform(curandGen, bFp32, numMatrixBDates));

        float *c;
        cudaErrCheck(hipMalloc((void **) &c, numMatrixCDates * sizeof(float)));
        curandErrCheck(hiprandGenerateUniform(curandGen, c, numMatrixCDates));

        cudaErrCheck(hipMemcpy(cCublas, c, numMatrixCDates, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaEx, c, numMatrixCDates, hipMemcpyDeviceToDevice));

        curandErrCheck(hiprandDestroyGenerator(curandGen));

        const int numThreadPerBlock = 256;
        convertFp32ToFp16<<< (numMatrixADates + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            aFp16, aFp32, numMatrixADates);
        convertFp32ToFp16<<< (numMatrixBDates + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            bFp16, bFp32, numMatrixBDates);
    }

    /* using cuBLAS computation */
    {
        printf("Running with cuBLAS...\n");

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));

        hipblasHandle_t cublasHandle;
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipEventRecord(startcublas));
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    MATRIX_M, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    aFp16, HIP_R_16F, MATRIX_M,
                                    bFp16, HIP_R_16F, MATRIX_K,
                                    &beta,
                                    cCublas, HIP_R_32F, MATRIX_M,
                                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));

        float cublasTime;
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
        printf("hipblasGemmEx time : %fms\n", cublasTime);

        cublasErrCheck(hipblasDestroy(cublasHandle));

        cudaErrCheck(hipEventDestroy(startcublas));
        cudaErrCheck(hipEventDestroy(stopcublas));
    }

    /* using wmmaExample computation */
    {
        printf("Running with wmmaExample...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        dim3 gridDim;
        dim3 blockDim;

        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmmaExample<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aFp16, bFp16, cWmmaEx);
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmmaExample time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    /* error checking */
    {
        printf("\nChecking results...\n");

        float *cCublasHost = (float *) malloc(numMatrixCDates * sizeof(float));
        float *cWmmaExHost = (float *) malloc(numMatrixCDates * sizeof(float));

        cudaErrCheck(hipMemcpy(cWmmaExHost, cWmmaEx, numMatrixCDates * sizeof(float), hipMemcpyDeviceToHost));
        cudaErrCheck(hipMemcpy(cCublasHost, cCublas, numMatrixCDates * sizeof(float), hipMemcpyDeviceToHost));

        int errors = 0;
        for (int idx = 0; idx < numMatrixCDates; ++idx) {
            float cublasRes = cCublasHost[idx];
            float wmmaExRes = cWmmaEx[idx];
            float diffDats = fabs(cublasRes - wmmaExRes);

            float relativeErr = diffDats / cublasRes;
            float eps = 1e-4;
            if (relativeErr >= eps) {
                ++errors;
                if (errors < 10) {
                    printf("error : cublasRes = %f, wmmaExRes = %f\n", cublasRes, wmmaExRes);
                }
            }
        }

        free(cCublasHost);
        free(cWmmaExHost);
    }

    cudaErrCheck(hipFree(aFp32));
    cudaErrCheck(hipFree(bFp32));
    cudaErrCheck(hipFree(aFp16));
    cudaErrCheck(hipFree(bFp16));
    cudaErrCheck(hipFree(cWmmaEx));
    cudaErrCheck(hipFree(cCublas));

    return 0;
}