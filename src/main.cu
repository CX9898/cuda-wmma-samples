#include "kernel.cuh"

int main() {
    float *aFp32;
    float *bFp32;

    half *aFp16;
    half *bFp16;

    float *cCublas;
    float *cWmmaEx;
    float *cWmmaEx2;

    const float alpha = 2.0f;
    const float beta = 2.0f;

    const int numMatrixADates = MATRIX_M * MATRIX_K;
    const int numMatrixBDates = MATRIX_K * MATRIX_N;
    const int numMatrixCDates = MATRIX_M * MATRIX_N;

    cudaErrCheck(hipMalloc((void **) &aFp32, numMatrixADates * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &bFp32, numMatrixBDates * sizeof(float)));

    cudaErrCheck(hipMalloc((void **) &aFp16, numMatrixADates * sizeof(half)));
    cudaErrCheck(hipMalloc((void **) &bFp16, numMatrixBDates * sizeof(half)));

    cudaErrCheck(hipMalloc((void **) &cCublas, numMatrixCDates * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &cWmmaEx, numMatrixCDates * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &cWmmaEx2, numMatrixCDates * sizeof(float)));

    /* using hiprand to initialize */
    {
        hiprandGenerator_t curandGen;

        curandErrCheck(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(curandGen, aFp32, numMatrixADates));
        curandErrCheck(hiprandGenerateUniform(curandGen, bFp32, numMatrixBDates));

        float *c;
        cudaErrCheck(hipMalloc((void **) &c, numMatrixCDates * sizeof(float)));
        curandErrCheck(hiprandGenerateUniform(curandGen, c, numMatrixCDates));

        cudaErrCheck(hipMemcpy(cCublas, c, numMatrixCDates, hipMemcpyDeviceToDevice));
        cudaErrCheck(hipMemcpy(cWmmaEx, c, numMatrixCDates, hipMemcpyDeviceToDevice));

        curandErrCheck(hiprandDestroyGenerator(curandGen));

        const int numThreadPerBlock = 256;
        convertFp32ToFp16<<< (numMatrixADates + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            aFp16, aFp32, numMatrixADates);
        convertFp32ToFp16<<< (numMatrixBDates + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            bFp16, bFp32, numMatrixBDates);
    }

    /* using cuBLAS computation */
    {
        printf("Running with cuBLAS...\n");

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));

        hipblasHandle_t cublasHandle;
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipEventRecord(startcublas));
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    MATRIX_M, MATRIX_N, MATRIX_K,
                                    &alpha,
                                    aFp16, HIP_R_16F, MATRIX_M,
                                    bFp16, HIP_R_16F, MATRIX_K,
                                    &beta,
                                    cCublas, HIP_R_32F, MATRIX_M,
                                    HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));

        float cublasTime;
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
        printf("hipblasGemmEx time : %fms\n", cublasTime);

        cublasErrCheck(hipblasDestroy(cublasHandle));

        cudaErrCheck(hipEventDestroy(startcublas));
        cudaErrCheck(hipEventDestroy(stopcublas));
    }

    /* using wmma-example computation */
    {
        printf("Running with wmma-example...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        dim3 gridDim;
        dim3 blockDim;

        // blockDim.x must be a multple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmma_example<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aFp16, bFp16, cWmmaEx2);
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmmaExample time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

    /* using wmmaExample computation */
    {
        printf("Running with wmmaExample...\n");

        hipEvent_t startWmmaEx;
        hipEvent_t stopWmmaEx;

        cudaErrCheck(hipEventCreate(&startWmmaEx));
        cudaErrCheck(hipEventCreate(&stopWmmaEx));

        dim3 gridDim;
        dim3 blockDim;

        cudaErrCheck(hipEventRecord(startWmmaEx));
        wmmaExample<<<gridDim, blockDim>>>(MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta, aFp16, bFp16, cWmmaEx);
        cudaErrCheck(hipEventRecord(stopWmmaEx));
        cudaErrCheck(hipEventSynchronize(stopWmmaEx));

        float wmmaTime;
        cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWmmaEx, stopWmmaEx));
        printf("wmmaExample time : %fms\n", wmmaTime);

        cudaErrCheck(hipEventDestroy(startWmmaEx));
        cudaErrCheck(hipEventDestroy(stopWmmaEx));
    }

//    if (!checkData(numMatrixCDates, cCublas, cWmmaEx)) {
//        fprintf(stderr,"The results of cublas and wmmaEx are inconsistent\n");
//    }
    if (!checkData(numMatrixCDates, cCublas, cWmmaEx2)) {
        fprintf(stderr,"The results of cublas and wmmaEx2 are inconsistent\n");
    }

    cudaErrCheck(hipFree(aFp32));
    cudaErrCheck(hipFree(bFp32));
    cudaErrCheck(hipFree(aFp16));
    cudaErrCheck(hipFree(bFp16));
    cudaErrCheck(hipFree(cCublas));
    cudaErrCheck(hipFree(cWmmaEx));
    cudaErrCheck(hipFree(cWmmaEx2));

    return 0;
}